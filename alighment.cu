#include "alighment.h"
__host__
void alighment_gpu(char* h_seq_a, char* h_seq_b, int size)
{
    char* d_seq_a, d_seq_b;
    unsigned long int bytes = sizeof(int) * size;
    hipMalloc((void**)d_seq_a, (size_t)bytes);
    hipMalloc((void**)d_seq_b, (size_t)bytes);
	hipMemcpy(d_seq_a, h_seq_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_seq_b, h_seq_b, bytes, hipMemcpyHostToDevice);
    
    
    //do the work
    
    
    hipFree((void*) d_seq_a);
    hipFree((void*) d_seq_b);
}
